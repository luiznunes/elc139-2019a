#include "hip/hip_runtime.h"
%%writefile wavecuda2.cu
#include <cstdlib>
#include <stdio.h>
#include <sys/time.h>
#include <math.h>
#include "wave.h"

__global__
void work(int width, unsigned char* pic) {

  for (int row = 0; row < width; row++) {
    for (int col = 0; col < width; col++) {
      float fx = col - 1024/2;
      float fy = row - 1024/2;
      float d = sqrtf( fx * fx + fy * fy );
      unsigned char color = (unsigned char) (120.0f + 127.0f *
                                        cos(d/10.0f - threadIdx.x/7.0f) /
                                        (d/50.0f + 1.0f));
      pic[threadIdx.x * width * width + row * width + col] = (unsigned char) color;
    }
  }
   
}

int main(int argc, char** argv) {

  if (argc != 3) {
    printf("ERRO: usar %s largura_frame num_frames\n", argv[0]);
    exit(-1);
  }
  int width = atoi(argv[1]);
  if (width < 100) {
    printf("ERRO: largura_frame deve ser maior igual a 100\n");
    exit(-1);
  }
  int frames = atoi(argv[2]);
  if (frames < 1) {
    printf("ERRO: num_frames deve ser pelo menos 1\n");
    exit(-1);
  }
  printf("CUDA computing %d frames of %d by %d picture\n", frames, width, width);
    
  // allocate picture array
  unsigned char* pic;
  hipMalloc(&pic, frames*width*width*sizeof(unsigned char));
    
  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  work<<<1, frames>>>(width, pic);
    
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
    
  // termina o tempo
  gettimeofday(&end, NULL);
  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  printf("CUDA compute time: %.4f s\n", runtime);
    
  hipFree(pic);
  return 0;

}